#include "hip/hip_runtime.h"
#include "baseutils.cuh"

using namespace std;

// Use float to obtain the highest speed
__device__ float CosineSimilarity(const float *vec1, const float *vec2, int dim) {
  double dot = 0.0, denom_a = 0.0, denom_b = 0.0;
  for (int idx = 0; idx < dim; ++idx) {
    dot += vec1[idx] * vec2[idx];
    denom_a += vec1[idx] * vec1[idx];
    denom_b += vec2[idx] * vec2[idx];
  }
  return dot / (sqrt(denom_a) * sqrt(denom_b));
}

__global__ void CosineSimilarityKernel(const float *vecs1, const float *vecs2, float *result, unsigned int rows1, unsigned int rows2, unsigned int cols) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < rows1 * rows2) {
    int idx = index / rows2;
    int idy = index % rows2;
    result[index] = CosineSimilarity(vecs1 + idx * cols, vecs2 + idy * cols, cols);
  }
}

__global__ void QueryMax(const float *sim_matrix, unsigned int *result_array, unsigned int rows1, unsigned int rows2) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows1) {
		float max = 0.0;
		int max_index = 0;
		for (int idy = 0; idy < rows2; ++idy) {
			if (sim_matrix[index * rows2 + idy] > max) {
				max = sim_matrix[index * rows2 + idy];
				max_index = idy;
			}
		}
		result_array[index] = max_index;
	}
}

__global__ void QueryMin(const float *sim_matrix, unsigned int *result_array, unsigned int rows1, unsigned int rows2) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows1) {
		float min = 1.0;
		int min_index = 0;
		for (int idy = 0; idy < rows2; ++idy) {
			if (sim_matrix[index * rows2 + idy] < min) {
				min = sim_matrix[index * rows2 + idy];
				min_index = idy;
			}
		}
		result_array[index] = min_index;
	}
}



// Compute the cosine similarity between two batches of vectors
// Pure C++ function to do batch similarity computation
void CosineSimilarityBatch(const float *vecs1, const float *vecs2, float *result, int rows1, int rows2, int cols) {
  unsigned long int total = rows1 * rows2;
  int threads = 256;
  int blocks = (total + threads - 1) / threads;
  float *vecs1_device, *vecs2_device, *result_device;
  hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
  hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));
  hipMalloc(&result_device, total * sizeof(float));

  hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(result_device, result, total * sizeof(float), hipMemcpyHostToDevice);

  CosineSimilarityKernel<<<blocks, threads>>>(vecs1_device, vecs2_device, result_device, rows1, rows2, cols);

  hipMemcpy(result, result_device, total * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(vecs1_device);
  hipFree(vecs2_device);
  hipFree(result_device);
}

void CosineSimilarityQueryMax(const float *vecs1, const float *vecs2, unsigned int *ret_indexes, unsigned int rows1, unsigned int rows2, unsigned int cols) {
 unsigned long int total = rows1 * rows2;
 int threads = 256;
 int blocks = (total + threads - 1) / threads;

 unsigned int *result_device;  // cuda variable to host the M*N matrix results
 hipMalloc(&result_device, rows1 * sizeof(unsigned int));

 float *vecs1_device, *vecs2_device, *sim_matrix_device;  // cuda variable to host the M*X or N*X cooridnate results
 hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
 hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));
 hipMalloc(&sim_matrix_device, total * sizeof(float));   // No need for output the result

 hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
 CosineSimilarityKernel<<<blocks, threads>>>(vecs1_device, vecs2_device, sim_matrix_device, rows1, rows2, cols);  // Compute the cosine similarity
 QueryMax<<<blocks, threads>>>(sim_matrix_device, result_device, rows1, rows2);

 hipMemcpy(ret_indexes, result_device, rows1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
 hipFree(vecs1_device);
 hipFree(vecs2_device);
 hipFree(result_device);
 hipFree(sim_matrix_device);
}


void CosineSimilarityQueryMin(const float *vecs1, const float *vecs2, unsigned int *ret_indexes, unsigned int rows1, unsigned int rows2, unsigned int cols) {
 unsigned long int total = rows1 * rows2;
 int threads = 256;
 int blocks = (total + threads - 1) / threads;

 unsigned int *result_device;  // cuda variable to host the M*N matrix results
 hipMalloc(&result_device, rows1 * sizeof(unsigned int));

 float *vecs1_device, *vecs2_device, *sim_matrix_device;  // cuda variable to host the M*X or N*X cooridnate results
 hipMalloc(&sim_matrix_device, total * sizeof(float));   // No need for output the result
 hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
 hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));

 hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
 CosineSimilarityKernel<<<blocks, threads>>>(vecs1_device, vecs2_device, sim_matrix_device, rows1, rows2, cols);  // Compute the cosine similarity
 QueryMin<<<blocks, threads>>>(sim_matrix_device, result_device, rows1, rows2);

 hipMemcpy(ret_indexes, result_device, rows1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
 hipFree(vecs1_device);
 hipFree(vecs2_device);
 hipFree(result_device);
 hipFree(sim_matrix_device);
}
