#include "hip/hip_runtime.h"
#include "gpuutils.cuh"

__global__ void cosine_similarity_global(const float *vecs1, const float *vecs2, float *result, unsigned int rows1, unsigned int rows2, unsigned int cols) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < rows1 * rows2) {
    int idx = index / rows2;
    int idy = index % rows2;
    result[index] = cosine_similarity(vecs1 + idx * cols, vecs2 + idy * cols, cols);
  }
}

__global__ void QueryMax(const float *sim_matrix, unsigned int *result_array, unsigned int rows1, unsigned int rows2) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows1) {
		float max = 0.0;
		int max_index = 0;
		for (int idy = 0; idy < rows2; ++idy) {
			if (sim_matrix[index * rows2 + idy] > max) {
				max = sim_matrix[index * rows2 + idy];
				max_index = idy;
			}
		}
		result_array[index] = max_index;
	}
}

__global__ void QueryMin(const float *sim_matrix, unsigned int *result_array, unsigned int rows1, unsigned int rows2) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows1) {
		float min = 1.0;
		int min_index = 0;
		for (int idy = 0; idy < rows2; ++idy) {
			if (sim_matrix[index * rows2 + idy] < min) {
				min = sim_matrix[index * rows2 + idy];
				min_index = idy;
			}
		}
		result_array[index] = min_index;
	}
}



void CosineSimilarityBatch(const float *vecs1, const float *vecs2, float *result, int rows1, int rows2, int cols) {
  unsigned long int total = rows1 * rows2;
  int blocks = (total + BLOCK_SIZE - 1) / BLOCK_SIZE;
  float *vecs1_device, *vecs2_device, *result_device;
  hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
  hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));
  hipMalloc(&result_device, total * sizeof(float));

  hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(result_device, result, total * sizeof(float), hipMemcpyHostToDevice);

  cosine_similarity_global<<<blocks, BLOCK_SIZE>>>(vecs1_device, vecs2_device, result_device, rows1, rows2, cols);

  hipMemcpy(result, result_device, total * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(vecs1_device);
  hipFree(vecs2_device);
  hipFree(result_device);
}

void CosineSimilarityQueryMax(const float *vecs1, const float *vecs2, unsigned int *ret_indexes, unsigned int rows1, unsigned int rows2, unsigned int cols) {
  unsigned long int total = rows1 * rows2;
  int blocks = (total + BLOCK_SIZE - 1) / BLOCK_SIZE;

  unsigned int *result_device;  // cuda variable to host the M*N matrix results
  hipMalloc(&result_device, rows1 * sizeof(unsigned int));

  float *vecs1_device, *vecs2_device, *sim_matrix_device;  // cuda variable to host the M*X or N*X cooridnate results
  hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
  hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));
  hipMalloc(&sim_matrix_device, total * sizeof(float));   // No need for output the result

  hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
  cosine_similarity_global<<<blocks, BLOCK_SIZE>>>(vecs1_device, vecs2_device, sim_matrix_device, rows1, rows2, cols);  // Compute the cosine similarity
  QueryMax<<<blocks, BLOCK_SIZE>>>(sim_matrix_device, result_device, rows1, rows2);

  hipMemcpy(ret_indexes, result_device, rows1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipFree(vecs1_device);
  hipFree(vecs2_device);
  hipFree(result_device);
  hipFree(sim_matrix_device);
}


void CosineSimilarityQueryMin(const float *vecs1, const float *vecs2, unsigned int *ret_indexes, unsigned int rows1, unsigned int rows2, unsigned int cols) {
  unsigned long int total = rows1 * rows2;
  int blocks = (total + BLOCK_SIZE - 1) / BLOCK_SIZE;

  unsigned int *result_device;  // cuda variable to host the M*N matrix results
  hipMalloc(&result_device, rows1 * sizeof(unsigned int));

  float *vecs1_device, *vecs2_device, *sim_matrix_device;  // cuda variable to host the M*X or N*X cooridnate results
  hipMalloc(&sim_matrix_device, total * sizeof(float));   // No need for output the result
  hipMalloc(&vecs1_device, rows1 * cols * sizeof(float));
  hipMalloc(&vecs2_device, rows2 * cols * sizeof(float));

  hipMemcpy(vecs1_device, vecs1, rows1 * cols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(vecs2_device, vecs2, rows2 * cols * sizeof(float), hipMemcpyHostToDevice);
  cosine_similarity_global<<<blocks, BLOCK_SIZE>>>(vecs1_device, vecs2_device, sim_matrix_device, rows1, rows2, cols);  // Compute the cosine similarity
  QueryMin<<<blocks, BLOCK_SIZE>>>(sim_matrix_device, result_device, rows1, rows2);

  hipMemcpy(ret_indexes, result_device, rows1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipFree(vecs1_device);
  hipFree(vecs2_device);
  hipFree(result_device);
  hipFree(sim_matrix_device);
}